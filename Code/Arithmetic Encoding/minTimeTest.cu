#include "hip/hip_runtime.h"
#include<stdio.h>

__global__ void probGPU(int *t){
	int bid=threadIdx.x;
	*t=bid;
}

int main(){
	int *test;
	int test2=5;
	clock_t begin2, end2;
	double time_spent2=0;
	begin2 = clock();
	
	hipMalloc((void **)&test,sizeof(int));
	hipMemcpy(test,&test2,sizeof(int),hipMemcpyHostToDevice);
	probGPU<<<1,1>>>(test);
	hipMemcpy(&test2,test,sizeof(int),hipMemcpyDeviceToHost);
	end2 = clock();
	time_spent2 += (double)(end2 - begin2) / CLOCKS_PER_SEC;
	printf("%d.. Time= %lf",test2,time_spent2);
	return 0;
}