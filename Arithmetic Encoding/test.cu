
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c){
	*c=*a+*b;
}

int main(){
	int i=5,j=10,res;
	int *dev_i,*dev_j,*dev_res;
	
	hipMalloc((void **)&dev_i,sizeof(int));
	hipMalloc((void **)&dev_j,sizeof(int));
	hipMalloc((void **)&dev_res,sizeof(int));
	
	hipMemcpy(dev_i,&i,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_j,&j,sizeof(int),hipMemcpyHostToDevice);
	
	add<<<1,1>>>(dev_i,dev_j,dev_res);
	
	hipMemcpy(&res,dev_res,sizeof(int),hipMemcpyDeviceToHost);
	
	printf("The result from CPU is %d\n\tFrom GPU is %d",(i+j),res);

	return 0;
}